
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>
#include <sys/time.h>

// ANSI color escape codes
#define RED "\x1B[31m"
#define GREEN "\x1B[32m"
#define RESET "\x1B[0m"

// Function to initialize the Ising model with a random initial state
void initialize(int *grid, int n) {
    for (int i = 0; i < n * n; ++i) {
        grid[i] = (rand() % 2) * 2 - 1; // +1 or -1
    }
}

__global__ void update(int *current, int *next, int n) {
    extern __shared__ int shared[];

    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = row * n + col;

    int sharedIdx = threadIdx.x * blockDim.y + threadIdx.y;

    if (row < n && col < n) {
        int top = ((row - 1 + n) % n) * n + col;
        int left = row * n + (col - 1 + n) % n;
        int center = idx;
        int down = ((row + 1) % n) * n + col;
        int right = row * n + (col + 1) % n;

        // Load necessary data into shared memory
        shared[sharedIdx] = current[center];
        shared[sharedIdx + 1] = current[top];
        shared[sharedIdx + 2] = current[left];
        shared[sharedIdx + 3] = current[down];
        shared[sharedIdx + 4] = current[right];

        __syncthreads();

        // Calculate sum using shared memory
        int sum = shared[sharedIdx] +
		shared[sharedIdx + 1] +
		shared[sharedIdx + 2] +
		shared[sharedIdx + 3] +
		shared[sharedIdx + 4];

        next[idx] = (sum > 0) ? 1 : -1;
    }
}

// Function to print the current state of the Ising model
void printState(int *grid, int n) {
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            if(grid[i * n + j] == 1) {
                printf(GREEN "■" RESET);
            } else {
                printf(RED "■" RESET);
            }
        }
        printf("\n");
    }
    printf("-----------------------------------------------\n");
}

int main(int argc, char *argv[]) {
    int seed = 42;
    srand(seed);

    struct timeval t1, t2;
    int n = atoi(argv[1]);
    int k = atoi(argv[2]);
    char c;

    if (argc == 4) {
        c = *argv[3];
    } else{
        c = '\0'; // Default block size
    }

    gettimeofday(&t1, 0);

    // Allocate memory for two grids (current and next states) on GPU
    int *device_grid1, *device_grid2;
    hipMalloc((void **)&device_grid1, n * n * sizeof(int));
    hipMalloc((void **)&device_grid2, n * n * sizeof(int));

    // Initialize the Ising model with a random initial state
    int *host_grid1 = (int *)malloc(n * n * sizeof(int));
    initialize(host_grid1, n);
    hipMemcpy(device_grid1, host_grid1, n * n * sizeof(int), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 blockDim(5, 5);
    dim3 gridDim((n + blockDim.x - 1) / blockDim.x, (n + blockDim.y - 1) / blockDim.y);

    // Simulation loop on GPU
    for (int i = 0; i < k; ++i) {
        // Perform one iteration of the Ising model
        update<<<gridDim, blockDim, 25 * sizeof(int)>>>(device_grid1, device_grid2, n);

        // Swap the pointers for the next iteration
        int *temp = device_grid1;
        device_grid1 = device_grid2;
        device_grid2 = temp;

	if(c == 'a'){
         	hipMemcpy(host_grid1, device_grid1, n * n * sizeof(int), hipMemcpyDeviceToHost);
         	printState(host_grid1, n);
	 	usleep(100000);
	}
    }

    // Print the final state of the Ising model (if needed)
    hipMemcpy(host_grid1, device_grid1, n * n * sizeof(int), hipMemcpyDeviceToHost);
    //printState(host_grid1, n);

    // Free allocated memory on GPU
    hipFree(device_grid1);
    hipFree(device_grid2);

    // Free allocated memory on CPU
    free(host_grid1);

    //HANDLE_ERROR(cudaThreadSynchronize());
    gettimeofday(&t2, 0);
    double time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;
    printf("Time to generate:  %3.1f ms \n", time);

    return 0;
}
