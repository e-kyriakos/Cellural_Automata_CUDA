
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>
#include <sys/time.h>

// ANSI color escape codes
#define RED "\x1B[31m"
#define GREEN "\x1B[32m"
#define RESET "\x1B[0m"

// Function to initialize the Ising model with a random initial state
void initialize(int *grid, int n) {
    for (int i = 0; i < n * n; ++i) {
        grid[i] = (rand() % 2) * 2 - 1; // +1 or -1
    }
}

// CUDA kernel for simulating one iteration of the Ising model
__global__ void update(int *current, int *next, int n) {
	int i = blockIdx.x;
	int j = blockIdx.y;

        // Calculate the new state based on the majority of neighbors
        int sum = current[((i - 1 + n) % n) * n + j] +
                  current[i * n + (j - 1 + n) % n] +
                  current[i * n + j] +
                  current[((i + 1) % n) * n + j] +
                  current[i * n + (j + 1) % n];

        next[i * n + j] = (sum > 0) ? 1 : -1;
}

// Function to print the current state of the Ising model
void printState(int *grid, int n) {
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
		if(grid[i * n + j] == 1){
	  		printf(GREEN "■" RESET);
		}
		else{
	  		printf(RED "■" RESET);
        	}
    	}
        printf("\n");
    }
    printf("-----------------------------------------------\n");
}

int main(int argc, char *argv[]) {

    int seed = 42;
    srand(seed);

    if (argc < 2 && argc > 3) {
        printf("Usage: %s <size_n> <iterations_k> <a>\n", argv[0]);
        return 1;
    }

    struct timeval t1, t2;
    int n = atoi(argv[1]);
    int k = atoi(argv[2]);
    char c;


    if (argc == 4) {
        c = *argv[3];
        printf("c = %c\n", c);
    } else {
        c = '\0';
    }

    gettimeofday(&t1, 0);

    // Allocate memory for two grids (current and next states) on GPU
    int *device_grid1, *device_grid2;
    hipMalloc((void **)&device_grid1, n * n * sizeof(int));
    hipMalloc((void **)&device_grid2, n * n * sizeof(int));

    // Initialize the Ising model with a random initial state
    int *host_grid1 = (int *)malloc(n * n * sizeof(int));
    initialize(host_grid1, n);
    hipMemcpy(device_grid1, host_grid1, n * n * sizeof(int), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 blockDim(1);
    dim3 gridDim(n, n);

    // Simulation loop on GPU
    for (int i = 0; i < k; ++i) {
        // Perform one iteration of the Ising model
        update<<<gridDim, blockDim>>>(device_grid1, device_grid2, n);

        // Swap the pointers for the next iteration
        int *temp = device_grid1;
        device_grid1 = device_grid2;
        device_grid2 = temp;

	// Write 'a' argument for animation of 10 fps
	if(c == 'a'){
         	hipMemcpy(host_grid1, device_grid1, n * n * sizeof(int), hipMemcpyDeviceToHost);
         	printState(host_grid1, n);
	 	usleep(100000);
	}
    }

    // Print the final state of the Ising model (if needed)
    hipMemcpy(host_grid1, device_grid1, n * n * sizeof(int), hipMemcpyDeviceToHost);
    //printState(host_grid1, n);

    // Free allocated memory on GPU
    hipFree(device_grid1);
    hipFree(device_grid2);

    // Free allocated memory on CPU
    free(host_grid1);

    //HANDLE_ERROR(cudaThreadSynchronize());
    gettimeofday(&t2, 0);
    double time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;
    printf("Time to generate:  %3.1f ms \n", time);

    return 0;
}
